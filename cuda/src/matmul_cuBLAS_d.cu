#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>

#define dim 10000
#define error 0.01
#define USECPSEC 1000000

void serial_mat_mult(double* mat_1, double* mat_2, double* ans_mat) {
    int i, j, k;
    for (i = 0;i < dim;i++) {
        for(j = 0;j < dim; j++) {
            ans_mat[i * dim + j] = 0.0;
            for(k = 0;k < dim; k++) {
                ans_mat[i * dim + j] += mat_1[i * dim + k] * mat_2[k * dim + j];
            }
        }
    }
    return;
}

int main() {
    //single precision matrix multiplication first
    // will benchmark later
    double *host_a, *host_b, *host_c;
    double *dev_a, *dev_b, *dev_c;

    host_a = (double*)malloc(dim * dim * sizeof(double));
    host_b = (double*)malloc(dim * dim * sizeof(double));
    host_c = (double*)malloc(dim * dim * sizeof(double));
    hipMalloc((void **)&dev_a, dim * dim * sizeof(double));
    hipMalloc((void **)&dev_b, dim * dim * sizeof(double));
    hipMalloc((void **)&dev_c, dim * dim * sizeof(double));

    srand((unsigned int) time(NULL)); int i; int j;
    for(i = 0; i < dim * dim; i++) {
        host_a[i] = (double)rand()/RAND_MAX;
        host_b[i] = (double)rand()/RAND_MAX;
    }

    hipMemcpy(dev_a, host_a, dim * dim * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, dim * dim * sizeof(double), hipMemcpyHostToDevice);

    hipblasHandle_t handle; hipblasCreate(&handle);
    double scalar_1 = 1.0f;
    double scalar_2 = 0.0f;
    struct timeval start, end, diff;

    hipEvent_t start_event, stop_event;
    float elapsed_time;

    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);

    hipEventRecord(start_event, 0);
    // cuBLAS stores arrays in column major, not row-major, order
    hipblasDgemm(handle, 
                HIPBLAS_OP_T, HIPBLAS_OP_T,
                dim, dim, dim,
                &scalar_1,
                dev_a, dim,
                dev_b, dim,
                &scalar_2, 
                dev_c, dim
    );
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time, start_event, stop_event);
    printf("cuBLAS took %f milliseconds \n", elapsed_time);

    hipMemcpy(host_c, dev_c, dim * dim * sizeof(double), hipMemcpyDeviceToHost);

    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);



    double* serial_ans = (double*)malloc(dim * dim * sizeof(double));
    gettimeofday(&start, NULL);
    serial_mat_mult(host_a, host_b, serial_ans);
    gettimeofday(&end, NULL);
    timersub(&end, &start, &diff);

    printf("serial code took %f seconds \n", diff.tv_sec + (double)diff.tv_usec / USECPSEC);

    for(i = 0;i < dim; i++) {
        for(j = 0; j < dim; j++) {
            // the result is transposed
            if(fabs(host_c[j * dim + i] - serial_ans[i * dim + j]) > error) {
                printf("There was a calculation error \n");
                printf("The error percent was %f \n", (fabs(host_c[i * dim + j] - serial_ans[i * dim + j]))/serial_ans[i * dim + j]);
                return 0;
            }
        }
    }
    printf("The calculation was correct \n");
    free(host_a); free(host_b); free(host_c); free(serial_ans);
    hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);

    return 0;
}
