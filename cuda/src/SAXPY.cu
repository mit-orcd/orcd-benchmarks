#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define dim 4096*4096
#define numThreads 1024

__global__ void solve(int *a, int *b, int *c, int s1, int s2) {
    int thd_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (thd_index < dim) {
        c[thd_index] = s1 * a[thd_index] + s2 * b[thd_index];
    }
    return;
}

void solve_serial(int *a, int *b, int *c, int s1, int s2) {
    // vectors fo dimension dim
    int i;
    for (i = 0;i < dim;i++) {
        c[i] = s1 * a[i] + s2 * b[i];
    }
    return;
}

int* randArr(int d) {
    int* ans = (int*)malloc(d * sizeof(int));
    int i;
    for (i = 0;i < d;i++) {
        ans[i] = 1 + rand() % 100;
    }
    return ans;
}

int main() {
    srand(time(NULL));
    int* host_a = randArr(dim);
    int* host_b = randArr(dim);
    int* host_c = (int*)malloc(dim * sizeof(int));

    int s1 = 1;
    int s2 = 2;
    struct timeval start, end, diff;

    int *dev_a, *dev_b, *dev_c;
    hipMalloc((void **)&dev_a, dim * sizeof(int));
    hipMalloc((void **)&dev_b, dim * sizeof(int));
    hipMalloc((void **)&dev_c, dim * sizeof(int));

    hipMemcpy(dev_a, host_a, dim * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, dim * sizeof(int), hipMemcpyHostToDevice);
    int numBlocks = (dim + numThreads - 1) / numThreads;

    gettimeofday(&start, NULL);
    solve<<<numBlocks, numThreads>>>(dev_a, dev_b, dev_c, s1, s2);
    hipDeviceSynchronize();
    gettimeofday(&end, NULL);
    timersub(&end, &start, &diff);
    printf("Cuda code took %.8f seconds \n", diff.tv_sec + (double)diff.tv_usec / 1000000);

    hipMemcpy(host_c, dev_c, dim * sizeof(int), hipMemcpyDeviceToHost);

    gettimeofday(&start, NULL);
    solve_serial(host_a, host_b, host_c, s1, s2);
    gettimeofday(&end, NULL);
    timersub(&end, &start, &diff);
    printf("Serial code took %.8f seconds \n", diff.tv_sec + (double)diff.tv_usec / 1000000);
    
    free(host_a); free(host_b); free(host_c);
    hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);

    return 0;

}
