#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#define dim 64
#define numThreads 32

__device__ int index(int r, int c) {
    return r * dim + c;
}

__device__ void add_rows(float* device_m, int r1, int r2, float scale) {
    // adds the entries from r1 * scale into r2
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dim) {
        device_m[index(r1, idx)] += scale * device_m[index(r2, idx)];
    }
    return;
}

__global__ void add1(float* device_m) {
    //adds 1 to every entry
    float ratio = -1 * device_m[index(1, 0)] / device_m[index(0, 0)];
    add_rows(device_m, 0, 1, ratio);
    return;
}

int main() {
    float* host_m = (float*)malloc(dim * dim * sizeof(float));
    int i, j;
    for(i = 0; i < dim; i++) {
        for(j = 0; j < dim; j++) {
            host_m[i * dim + j] = (float)(i + j + 1);
        }
    }
    float* device_m;

    hipMalloc((void**)&device_m, dim * dim * sizeof(float));
    hipMemcpy(device_m, host_m, dim * dim * sizeof(float), hipMemcpyHostToDevice);
    
    int blockSize = (dim + numThreads - 1) / numThreads;

    add1<<<blockSize, threadSize>>>(device_m);

    hipMemcpy(host_m, device_m, dim * dim * sizeof(float), hipMemcpyDeviceToHost);

    printf("The entry that should be made 0 is equal to: %f \n", host_m[dim]);

    free(host_m); hipFree(device_m);
    return 0;
}
