#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void sayHelloWorld(int *device_cnt) {
    printf("Hello World from Block %d and Thread %d \n", blockIdx.x, threadIdx.x);
    atomicAdd(device_cnt, 1);
    return;
}

#define N 16

int main() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int host_cnt = 0;
    int *device_cnt;
    hipMalloc((void **)&device_cnt, sizeof(int));
    hipMemcpy(device_cnt, &host_cnt, sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(start);
    sayHelloWorld<<<4, 4>>>(device_cnt);
    hipEventRecord(stop);
    hipMemcpy(&host_cnt, device_cnt, sizeof(int), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);

    printf("Number of cores: %d \n", host_cnt);

    hipDeviceSynchronize();
    return 0;
}
