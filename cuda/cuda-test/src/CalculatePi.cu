#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define intervals 5000000000LL
#define numThreads 1024
#define USECPSEC 1000000

__global__ void solve(double *device_sums) {
    __shared__ double shared_sum[numThreads];
    long long int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= intervals) {
        return;
    }
    double inc = 1.0 / intervals;
    double x = (index + 0.5) * inc;
    shared_sum[threadIdx.x] = 4.0 * inc / (x * x + 1.0);
    __syncthreads();
    int i;
    for(i = numThreads / 2; i > 0; i >>=1) {
        if (threadIdx.x < i) { // numThreads needs to be a power of 2 to be accurate!
            shared_sum[threadIdx.x] += shared_sum[threadIdx.x + i];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        device_sums[blockIdx.x] = shared_sum[0];
    }
    return;
}

double solve() {
    long long int i;
    double inc = 1.0 / intervals;
    double sum = 0.0;
    for(i = 0;i < intervals; i++) {
        double x = (i + 0.5) * inc;
        sum += 4.0 * inc / (x * x + 1.0);
    }
    return sum;
}


int main() {
    double *host_sums;
    double *device_sums;
    long long int numBlocks = (intervals + numThreads - 1) / numThreads;

    host_sums = (double*)malloc(numBlocks * sizeof(double));
    hipMalloc((void **)&device_sums, sizeof(double) * numBlocks);

    struct timeval start, end, diff;
    gettimeofday(&start, NULL);
    solve<<<numBlocks, numThreads>>>(device_sums);
    hipDeviceSynchronize();
    gettimeofday(&end, NULL);
    timersub(&end, &start, &diff);
    printf("Kernel execution time of this program is %.8f \n", diff.tv_sec + (double)diff.tv_usec / USECPSEC);
    hipMemcpy(host_sums, device_sums, numBlocks * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(device_sums);
    double cuda_approx = 0.0; int i;
    for (i = 0; i < numBlocks; i++) {
        cuda_approx += host_sums[i];
    }
    free(host_sums);
    printf("Cuda approximation for pi with %lld rectangles: %.15f \n", intervals, cuda_approx);

    gettimeofday(&start, NULL);
    double serial_sum = solve();
    gettimeofday(&end, NULL);
    timersub(&end, &start, &diff);
    printf("Serial execution time of this program is %.8f \n", diff.tv_sec + (double)diff.tv_usec / USECPSEC);
    printf("Serial approximation for pi with %lld rectangles: %.15f \n", intervals, serial_sum);
    return 0;
}
