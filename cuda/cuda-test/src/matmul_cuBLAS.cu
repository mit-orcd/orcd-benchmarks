#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>

#define dim 100
#define error 0.01
#define USECPSEC 1000000

void serial_mat_mult(float* mat_1, float* mat_2, float* ans_mat) {
    int i, j, k;
    for (i = 0;i < dim;i++) {
        for(j = 0;j < dim; j++) {
            ans_mat[i * dim + j] = 0.0;
            for(k = 0;k < dim; k++) {
                ans_mat[i * dim + j] += mat_1[i * dim + k] * mat_2[k * dim + j];
            }
        }
    }
    return;
}

int main() {
    //single precision matrix multiplication first
    // will benchmark later
    float *host_a, *host_b, *host_c;
    float *dev_a, *dev_b, *dev_c;

    host_a = (float*)malloc(dim * dim * sizeof(float));
    host_b = (float*)malloc(dim * dim * sizeof(float));
    host_c = (float*)malloc(dim * dim * sizeof(float));
    hipMalloc((void **)&dev_a, dim * dim * sizeof(float));
    hipMalloc((void **)&dev_b, dim * dim * sizeof(float));
    hipMalloc((void **)&dev_c, dim * dim * sizeof(float));

    srand((unsigned int) time(NULL)); int i; int j;
    for(i = 0; i < dim * dim; i++) {
        host_a[i] = (float)rand()/RAND_MAX;
        host_b[i] = (float)rand()/RAND_MAX;
    }

    hipMemcpy(dev_a, host_a, dim * dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, dim * dim * sizeof(float), hipMemcpyHostToDevice);

    hipblasHandle_t handle; hipblasCreate(&handle);
    float scalar_1 = 1.0f;
    float scalar_2 = 0.0f;
    struct timeval start, end, diff;

    hipEvent_t start_event, stop_event;
    float elapsed_time;

    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);

    hipEventRecord(start_event, 0);
    // cuBLAS stores arrays in column major, not row-major, order
    hipblasSgemm(handle, 
                HIPBLAS_OP_T, HIPBLAS_OP_T,
                dim, dim, dim,
                &scalar_1,
                dev_a, dim,
                dev_b, dim,
                &scalar_2, 
                dev_c, dim
    );
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time, start_event, stop_event);
    printf("cuBLAS took %f milliseconds \n", elapsed_time);

    hipMemcpy(host_c, dev_c, dim * dim * sizeof(float), hipMemcpyDeviceToHost);

    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);



    float* serial_ans = (float*)malloc(dim * dim * sizeof(float));
    gettimeofday(&start, NULL);
    serial_mat_mult(host_a, host_b, serial_ans);
    gettimeofday(&end, NULL);
    timersub(&end, &start, &diff);

    printf("serial code took %f seconds \n", diff.tv_sec + (double)diff.tv_usec / USECPSEC);

    for(i = 0;i < dim; i++) {
        for(j = 0; j < dim; j++) {
            // the result is transposed
            if(fabs(host_c[j * dim + i] - serial_ans[i * dim + j]) > error) {
                printf("There was a calculation error \n");
                printf("The error percent was %f \n", (fabs(host_c[i * dim + j] - serial_ans[i * dim + j]))/serial_ans[i * dim + j]);
                return 0;
            }
        }
    }
    printf("The calculation was correct \n");
    free(host_a); free(host_b); free(host_c); free(serial_ans);
    hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);

    return 0;
}
