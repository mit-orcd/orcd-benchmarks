#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define dim 10000
#define max_val 10
#define max_error 0.01
#define numThreads_x 16
#define numThreads_y 16

__global__ void mat_mult(float* dev_matrix, float* mat_1, float* mat_2) {
    /* Runs matrix multiplication
    * all matrices are flattened 2D matrices of size dim
    * should fill in values of dev_matrix
    */
    int i, j, k;
    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < dim && j < dim) {
        dev_matrix[i * dim + j] = 0.0;
        for(k = 0; k < dim; k++) {
            dev_matrix[i * dim + j] += mat_1[i * dim + k] * mat_2[k * dim + j];
        }
    }
    return;
}

void serial_mat_mult(float* ans_mat, float* mat_1, float* mat_2) {
    int i, j, k;
    for (i = 0;i < dim;i++) {
        for(j = 0;j < dim; j++) {
            ans_mat[i * dim + j] = 0.0;
            for(k = 0;k < dim; k++) {
                ans_mat[i * dim + j] += mat_1[i * dim + k] * mat_2[k * dim + j];
            }
        }
    }
    return;
}
// __host__ __device__ int matrixIndex(int r, int c) {
//     return r * dim + c;
// }

int main() {
    float *host_1, *host_2, *host_ans;
    float *dev_1, *dev_2, *dev_ans;
    struct timeval start, end, diff;

    host_1 = (float*)malloc(dim * dim * sizeof(float));
    host_2 = (float*)malloc(dim * dim * sizeof(float));
    host_ans = (float*)malloc(dim * dim * sizeof(float));
    hipMalloc((void**)&dev_1, dim * dim * sizeof(float));
    hipMalloc((void**)&dev_2, dim * dim * sizeof(float));
    hipMalloc((void**)&dev_ans, dim * dim * sizeof(float));

    srand(time(NULL));
    int i, j;
    for(i = 0;i < dim; i ++) {
        for(j = 0; j < dim; j++) {
            host_1[i * dim + j] = ((float)rand() / RAND_MAX) * max_val;
            host_2[i * dim + j] = ((float)rand() / RAND_MAX) * max_val;
        }
    }
    hipMemcpy(dev_1, host_1, dim * dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_2, host_2, dim * dim * sizeof(float), hipMemcpyHostToDevice);
    
    int numBlocks_x = (dim + numThreads_x - 1) / numThreads_x;
    int numBlocks_y = (dim + numThreads_y - 1) / numThreads_y;
    dim3 gridSize(numBlocks_x, numBlocks_y);
    dim3 threadSize(numThreads_x, numThreads_y);
    
    gettimeofday(&start, NULL);
    mat_mult<<<gridSize, threadSize>>>(dev_ans, dev_1, dev_2);
    hipDeviceSynchronize();
    gettimeofday(&end, NULL);
    timersub(&end, &start, &diff);
    printf("Cuda code took %.8f seconds \n", diff.tv_sec + (double)diff.tv_usec / 1000000);

    hipMemcpy(host_ans, dev_ans, dim * dim * sizeof(float), hipMemcpyDeviceToHost);

    float* serial_ans = (float*)malloc(dim * dim * sizeof(float));
    gettimeofday(&start, NULL);
    serial_mat_mult(serial_ans, host_1, host_2);
    gettimeofday(&end, NULL);
    timersub(&end, &start, &diff);

    printf("Serial code took %.8f seconds \n", diff.tv_sec + (double)diff.tv_usec / 1000000);

    for(i = 0;i < dim; i++) {
        for(j = 0; j < dim; j++) {
            if(fabs(host_ans[i * dim + j] - serial_ans[i * dim + j]) > max_error) {
                printf("There was a calculation error \n");
                printf("host got: %f \n", host_ans[i * dim + j]);
                printf("serial got: %f \n", serial_ans[i * dim + j]);
                return 0;
            }
        }
    }
    printf("The calculation was correct \n");

    free(host_1); free(host_2); free(host_ans); free(serial_ans);
    hipFree(dev_1); hipFree(dev_2); hipFree(dev_ans);
    
    return 0;
}
