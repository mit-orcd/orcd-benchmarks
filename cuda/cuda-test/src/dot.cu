#include "hip/hip_runtime.h"
//takes the dot product of two vectors a specified number of times
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define dim 1000000000
// when dim <= 100 million, the array is cached, leading to very fast cpu runtimes
// at this dimension, the cpu must access from lower memory levels, leading to poor performance
// the gpu has to read from global memory anyways, making this a fairer comparison
// when cpu can access cached memory speedup ~ 30x for gpu binary 
#define numThreads 32
#define USECPSEC 1000000


// __device__ void reduce(volatile double* blockSum, int tid) {
//     blockSum[tid] += blockSum[tid + 32];
//     blockSum[tid] += blockSum[tid + 16];
//     blockSum[tid] += blockSum[tid + 8];
//     blockSum[tid] += blockSum[tid + 4];
//     blockSum[tid] += blockSum[tid + 2];
//     blockSum[tid] += blockSum[tid + 1];
// }

// __global__ void solve_reduce_full_unroll(double* device_u, double* device_v, double* device_a) {
//     // Implementation of the final optimized version from Nvidia 
//     // didn't use the template for blockSize - perhaps can implement that optimization 
//     // assumed 1024 threads
//     __shared__ double blockSum[1024];
//     int thd_id = threadIdx.x + blockIdx.x * blockDim.x * 2;
//     if (thd_id < dim) {
//         blockSum[threadIdx.x] = device_u[thd_id] * device_v[thd_id];
//     }
//     if (thd_id + blockDim.x < dim) {
//         blockSum[threadIdx.x] += device_u[blockDim.x + thd_id] * device_v[blockDim.x + thd_id];
//     }
//     __syncthreads();
//     if (threadIdx.x < 512) {
//         blockSum[threadIdx.x] +=blockSum[threadIdx.x + 512]; 
//     }
//     __syncthreads();
//     if (threadIdx.x < 256) {
//         blockSum[threadIdx.x] +=blockSum[threadIdx.x + 256]; 
//     }
//     __syncthreads();
//     if (threadIdx.x < 128) {
//         blockSum[threadIdx.x] +=blockSum[threadIdx.x + 128]; 
//     }
//     __syncthreads();

//     if (threadIdx.x < 64) {
//         blockSum[threadIdx.x] +=blockSum[threadIdx.x + 64]; 
//     }
//     __syncthreads();

//     if (threadIdx.x < 32) {
//         reduce(blockSum, threadIdx.x);
//     }
//     if (threadIdx.x == 0) {
//         atomicAdd(device_a, blockSum[0]);
//     }
// }


__global__ void solve_binary_double_load(double* device_u, double* device_v, double* device_a) {
    // first reduces witin each block, then uses atomicAdd
    __shared__ double blockSum[numThreads];
    int thd_id = threadIdx.x + blockIdx.x * blockDim.x * 2;
    if (thd_id < dim) {
        blockSum[threadIdx.x] = device_u[thd_id] * device_v[thd_id];
    }
    if (thd_id + blockDim.x < dim) {
        blockSum[threadIdx.x] += device_u[blockDim.x + thd_id] * device_v[blockDim.x + thd_id];
    }
    __syncthreads();
    int offset;
    for(offset = numThreads / 2; offset >= 1; offset >>=1) {
        if (threadIdx.x < offset) {
            blockSum[threadIdx.x] += blockSum[threadIdx.x + offset];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        atomicAdd(device_a, blockSum[0]);
    }
    return;
}

__global__ void solve_binary(double* device_u, double* device_v, double* device_a) {
    // first reduces witin each block, then uses atomicAdd
    __shared__ double blockSum[numThreads];
    int thd_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (thd_id >= dim) {
        return;
    }
    blockSum[threadIdx.x] = device_u[thd_id] * device_v[thd_id];
    __syncthreads();
    int offset;
    for(offset = numThreads / 2; offset >= 1; offset >>=1) {
        if (threadIdx.x < offset) {
            blockSum[threadIdx.x] += blockSum[threadIdx.x + offset];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        atomicAdd(device_a, blockSum[0]);
    }
    return;
}

__global__ void solve_atomic(double* device_u, double* device_v, double* device_a) {
    //uses atomicAdd to add every part of the dot product
    int thd_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (thd_id < dim) {
        atomicAdd(device_a, device_u[thd_id] * device_v[thd_id]);
    }
    return;
}



double dot_serial(double* v1, double* v2) {
    int i;
    double ans = 0.0;
    for(i = 0; i < dim; i++ ) {
        ans += v1[i] * v2[i];
    }
    return ans;
}

int main() {

    double *host_v, *host_u;
    double *device_v, *device_u;
    double host_a, *device_a;
    struct timeval start, end, diff;
    host_u = (double*)malloc(dim * sizeof(double));
    host_v = (double*)malloc(dim * sizeof(double));
    if (host_u == NULL) {
        printf("Failed to allocate memory \n");
    }
    hipMalloc((void**)&device_v, dim * sizeof(double));
    hipMalloc((void**)&device_u, dim * sizeof(double));
    hipMalloc((void**)&device_a, sizeof(double));
    host_a = 0.0f; 

    int i; srand((unsigned int) time(NULL));
    for(i = 0;i < dim;i++) {
        host_v[i] = (double)rand() / RAND_MAX;
        host_u[i] = (double)rand() / RAND_MAX;
    }

    gettimeofday(&start, NULL);
    double exp = dot_serial(host_v, host_u);
    gettimeofday(&end, NULL);
    timersub(&end, &start, &diff);
    printf("The dot product is %f \n", exp);
    printf("Serial code took %f seconds to run \n", diff.tv_sec + (double)diff.tv_usec / USECPSEC);


    int blocks = (dim + numThreads - 1) / numThreads;
    hipMemcpy(device_v, host_v, dim * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_u, host_u, dim * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_a, &host_a, sizeof(double), hipMemcpyHostToDevice);

    gettimeofday(&start, NULL);
    solve_atomic<<<blocks, numThreads>>>(device_u, device_v, device_a);
    hipDeviceSynchronize();
    hipMemcpy(&host_a, device_a, sizeof(double), hipMemcpyDeviceToHost);
    gettimeofday(&end, NULL);
    timersub(&end, &start, &diff);

    printf("Cuda atomicAdd got %f for the dot product \n", host_a);
    printf("Cuda atomicAdd took %f time to run \n", diff.tv_sec + (double)diff.tv_usec / USECPSEC);

    hipMalloc((void**)&device_a, sizeof(double));

    gettimeofday(&start, NULL);
    solve_binary<<<blocks, numThreads>>>(device_u, device_v, device_a);
    hipDeviceSynchronize();
    hipMemcpy(&host_a, device_a, sizeof(double), hipMemcpyDeviceToHost);
    gettimeofday(&end, NULL);
    timersub(&end, &start, &diff);

    printf("Cuda atomicAdd with binary reduction got %f for the dot product \n", host_a);
    printf("Cuda atomicAdd with binary reduction took %f time to run \n", diff.tv_sec + (double)diff.tv_usec / USECPSEC);

    hipMalloc((void**)&device_a, sizeof(double));
    blocks /= 2;

    gettimeofday(&start, NULL);
    solve_binary_double_load<<<blocks, numThreads>>>(device_u, device_v, device_a);
    hipDeviceSynchronize();
    hipMemcpy(&host_a, device_a, sizeof(double), hipMemcpyDeviceToHost);
    gettimeofday(&end, NULL);
    timersub(&end, &start, &diff);

    printf("Cuda atomicAdd with binary reduction, double load got %f for the dot product \n", host_a);
    printf("Cuda atomicAdd with binary reduction, double load took %f time to run \n", diff.tv_sec + (double)diff.tv_usec / USECPSEC);


    // hipMalloc((void**)&device_a, sizeof(double));
    // host_a = 0.0;

    // gettimeofday(&start, NULL);
    // solve_reduce_full_unroll<<<blocks, numThreads>>>(device_u, device_v, device_a);
    // hipDeviceSynchronize();
    // hipMemcpy(&host_a, device_a, sizeof(double), hipMemcpyDeviceToHost);
    // gettimeofday(&end, NULL);
    // timersub(&end, &start, &diff);

    // printf("Cuda atomicAdd with binary reduction with loop unrolling got %f for the dot product \n", host_a);
    // printf("Cuda atomicAdd with binary reduction with loop unrolling optimization took %f time to run \n", diff.tv_sec + (double)diff.tv_usec / USECPSEC);

    free(host_v); free(host_u); 
    hipFree(device_v); hipFree(device_u);

    return 0;
    

}